#include "hip/hip_runtime.h"
#include <iostream>
#include <utils_VkFFT.h>
#include <hipfft/hipfftw.h>
#include <chrono>

struct interfaceFFTPlan {
    VkFFTConfiguration *config;
    VkFFTApplication *app;
    bool isBaked;
    bool notInit;
    VkFFTLaunchParams *lParams;
    hipDevice_t device;
    hipCtx_t context;
    int dataType;
    int device_id;
    uint64_t inputBufferSize;
    uint64_t outputBufferSize;
};

typedef enum vkfft_transform_dir {
    VKFFT_FORWARD_TRANSFORM = -1,
    VKFFT_BACKWARD_TRANSFORM = 1
} vkfft_transform_dir;

typedef struct interfaceFFTPlan interfaceFFTPlan;


VkFFTResult
performVulkanFFT(interfaceFFTPlan *plan, vkfft_transform_dir inverse, uint64_t num_iter) {
    VkFFTResult resFFT = VKFFT_SUCCESS;
    hipError_t res = hipSuccess;
    std::chrono::steady_clock::time_point timeSubmit = std::chrono::steady_clock::now();
    for (uint64_t i = 0; i < num_iter; i++) {
        resFFT = VkFFTAppend(plan->app, inverse, plan->lParams);
        if (resFFT != VKFFT_SUCCESS) return resFFT;
    }
    res = hipDeviceSynchronize();
    if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_SYNCHRONIZE;
    std::chrono::steady_clock::time_point timeEnd = std::chrono::steady_clock::now();
    double totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001;
    std::cout << totTime << std::endl;
    return resFFT;
}

VkFFTResult
performVulkanFFTiFFT(interfaceFFTPlan *plan, uint64_t num_iter) {
    VkFFTResult resFFT = VKFFT_SUCCESS;

    hipError_t res = hipSuccess;
    std::chrono::steady_clock::time_point timeSubmit = std::chrono::steady_clock::now();
    for (uint64_t i = 0; i < num_iter; i++) {
        resFFT = VkFFTAppend(plan->app, vkfft_transform_dir::VKFFT_FORWARD_TRANSFORM, plan->lParams);
        if (resFFT != VKFFT_SUCCESS) return resFFT;
        resFFT = VkFFTAppend(plan->app, vkfft_transform_dir::VKFFT_BACKWARD_TRANSFORM, plan->lParams);
        if (resFFT != VKFFT_SUCCESS) return resFFT;
    }
    res = hipDeviceSynchronize();
    if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_SYNCHRONIZE;
    std::chrono::steady_clock::time_point timeEnd = std::chrono::steady_clock::now();
    double totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001;
    std::cout << totTime << std::endl;

    return resFFT;
}

VkFFTResult get_VkFFT_double(VkGPU *vkGPU) {
    VkFFTResult resFFT = VKFFT_SUCCESS;

    hipError_t res = hipSuccess;

//    fftw_complex *inputC;
//    fftw_complex *inputC_double;
//    uint64_t dims[3] = {benchmark_dimensions[n][0], benchmark_dimensions[n][1], benchmark_dimensions[n][2]};
//
//    inputC = (fftw_complex *) (malloc(sizeof(fftw_complex) * dims[0] * dims[1] * dims[2]));
//    if (!inputC) return VKFFT_ERROR_MALLOC_FAILED;
//    inputC_double = (fftw_complex *) (malloc(sizeof(fftw_complex) * dims[0] * dims[1] * dims[2]));
//    if (!inputC_double) return VKFFT_ERROR_MALLOC_FAILED;
//    for (uint64_t l = 0; l < dims[2]; l++) {
//        for (uint64_t j = 0; j < dims[1]; j++) {
//            for (uint64_t i = 0; i < dims[0]; i++) {
//                inputC[i + j * dims[0] + l * dims[0] * dims[1]][0] = (double) (
//                        2 * ((double) rand()) / RAND_MAX - 1.0);
//                inputC[i + j * dims[0] + l * dims[0] * dims[1]][1] = (double) (
//                        2 * ((double) rand()) / RAND_MAX - 1.0);
//                inputC_double[i + j * dims[0] + l * dims[0] * dims[1]][0] = (double) inputC[i + j * dims[0] +
//                                                                                            l * dims[0] *
//                                                                                            dims[1]][0];
//                inputC_double[i + j * dims[0] + l * dims[0] * dims[1]][1] = (double) inputC[i + j * dims[0] +
//                                                                                            l * dims[0] *
//                                                                                            dims[1]][1];
//            }
//        }
//    }
//
//    fftw_plan p;
//
//    fftw_complex *output_FFTW = (fftw_complex *) (malloc(sizeof(fftw_complex) * dims[0] * dims[1] * dims[2]));
//    if (!output_FFTW) return VKFFT_ERROR_MALLOC_FAILED;
//    switch (benchmark_dimensions[n][3]) {
//        case 1:
//            p = fftw_plan_dft_1d((int) benchmark_dimensions[n][0], inputC_double, output_FFTW, -1,
//                                 FFTW_ESTIMATE);
//            break;
//        case 2:
//            p = fftw_plan_dft_2d((int) benchmark_dimensions[n][1], (int) benchmark_dimensions[n][0],
//                                 inputC_double, output_FFTW, -1, FFTW_ESTIMATE);
//            break;
//        case 3:
//            p = fftw_plan_dft_3d((int) benchmark_dimensions[n][2], (int) benchmark_dimensions[n][1],
//                                 (int) benchmark_dimensions[n][0], inputC_double, output_FFTW, -1,
//                                 FFTW_ESTIMATE);
//            break;
//    }
//
//    fftw_execute(p);

    float totTime = 0;
    int num_iter = 1;

    //VkFFT part

    VkFFTConfiguration configuration = {};
    VkFFTApplication app = {};

    configuration.FFTdim = benchmark_dimensions[n][3]; //FFT dimension, 1D, 2D or 3D (default 1).
    configuration.size[0] = benchmark_dimensions[n][0]; //Multidimensional FFT dimensions sizes (default 1). For best performance (and stability), order dimensions in descendant size order as: x>y>z.
    configuration.size[1] = benchmark_dimensions[n][1];
    configuration.size[2] = benchmark_dimensions[n][2];

    //After this, configuration file contains pointers to Vulkan objects needed to work with the GPU: VkDevice* device - created device, [uint64_t *bufferSize, VkBuffer *buffer, VkDeviceMemory* bufferDeviceMemory] - allocated GPU memory FFT is performed on. [uint64_t *kernelSize, VkBuffer *kernel, VkDeviceMemory* kernelDeviceMemory] - allocated GPU memory, where kernel for convolution is stored.
    configuration.device = &vkGPU->device;
    configuration.doublePrecision = true;

    uint64_t numBuf = 1;

    //Allocate buffers for the input data. - we use 4 in this example
    uint64_t *bufferSize = (uint64_t *) malloc(sizeof(uint64_t) * numBuf);
    if (!bufferSize) return VKFFT_ERROR_MALLOC_FAILED;
    for (uint64_t i = 0; i < numBuf; i++) {
        bufferSize[i] = {};
        bufferSize[i] = (uint64_t) sizeof(double) * 2 * configuration.size[0] * configuration.size[1] *
                        configuration.size[2] / numBuf;
    }

    hipFloatComplex *buffer = 0;

    for (uint64_t i = 0; i < numBuf; i++) {

        res = hipMalloc((void **) &buffer, bufferSize[i]);
        if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_ALLOCATE;

    }

    configuration.bufferNum = numBuf;
    // Can specify buffers at launch
    configuration.bufferSize = bufferSize;

    //Sample buffer transfer tool. Uses staging buffer of the same size as destination buffer, which can be reduced if transfer is done sequentially in small buffers.
    uint64_t shift = 0;
    for (uint64_t i = 0; i < numBuf; i++) {

        res = hipMemcpy(buffer, inputC, bufferSize[i], hipMemcpyHostToDevice);
        if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_COPY;

        shift += bufferSize[i];
    }
    //Initialize applications. This function loads shaders, creates pipeline and configures FFT based on configuration file. No buffer allocations inside VkFFT library.
    resFFT = initializeVkFFT(&app, configuration);
    if (resFFT != VKFFT_SUCCESS) return resFFT;
    //Submit FFT+iFFT.
    //num_iter = 1;
    //specify buffers at launch
    VkFFTLaunchParams launchParams = {};

    launchParams.buffer = (void **) &buffer;

    resFFT = performVulkanFFT(vkGPU, &app, &launchParams, -1, num_iter);
    if (resFFT != VKFFT_SUCCESS) return resFFT;
    fftw_complex *output_VkFFT = (fftw_complex *) (malloc(sizeof(fftw_complex) * dims[0] * dims[1] * dims[2]));
    if (!output_VkFFT) return VKFFT_ERROR_MALLOC_FAILED;
    //Transfer data from GPU using staging buffer.
    shift = 0;
    for (uint64_t i = 0; i < numBuf; i++) {

        res = hipMemcpy(output_VkFFT, buffer, bufferSize[i], hipMemcpyDeviceToHost);
        if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_COPY;

        shift += bufferSize[i];
    }
    double avg_difference[2] = {0, 0};
    double max_difference[2] = {0, 0};
    double avg_eps[2] = {0, 0};
    double max_eps[2] = {0, 0};
    for (uint64_t l = 0; l < dims[2]; l++) {
        for (uint64_t j = 0; j < dims[1]; j++) {
            for (uint64_t i = 0; i < dims[0]; i++) {
                uint64_t loc_i = i;
                uint64_t loc_j = j;
                uint64_t loc_l = l;

                double current_data_norm = sqrt(output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][0] *
                                                output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][0] +
                                                output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][1] *
                                                output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][1]);

                double current_diff_x_VkFFT = (
                        output_VkFFT[loc_i + loc_j * dims[0] + loc_l * dims[0] * dims[1]][0] -
                        output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][0]);
                double current_diff_y_VkFFT = (
                        output_VkFFT[loc_i + loc_j * dims[0] + loc_l * dims[0] * dims[1]][1] -
                        output_FFTW[i + j * dims[0] + l * dims[0] * dims[1]][1]);
                double current_diff_norm_VkFFT = sqrt(current_diff_x_VkFFT * current_diff_x_VkFFT +
                                                      current_diff_y_VkFFT * current_diff_y_VkFFT);
                if (current_diff_norm_VkFFT > max_difference[1]) max_difference[1] = current_diff_norm_VkFFT;
                avg_difference[1] += current_diff_norm_VkFFT;
                if ((current_diff_norm_VkFFT / current_data_norm > max_eps[1]) && (current_data_norm > 1e-16)) {
                    max_eps[1] = current_diff_norm_VkFFT / current_data_norm;
                }
                avg_eps[1] += (current_data_norm > 1e-10) ? current_diff_norm_VkFFT / current_data_norm : 0;
            }
        }
    }
    avg_difference[0] /= (dims[0] * dims[1] * dims[2]);
    avg_eps[0] /= (dims[0] * dims[1] * dims[2]);
    avg_difference[1] /= (dims[0] * dims[1] * dims[2]);
    avg_eps[1] /= (dims[0] * dims[1] * dims[2]);
    printf("VkFFT System: %" PRIu64 "x%" PRIu64 "x%" PRIu64 " avg_difference: %.15f max_difference: %.15f avg_eps: %.15f max_eps: %.15f\n",
           dims[0], dims[1], dims[2], avg_difference[1], max_difference[1], avg_eps[1], max_eps[1]);
    free(output_VkFFT);
    for (uint64_t i = 0; i < numBuf; i++) {
        hipFree(buffer);
    }
    deleteVkFFT(&app);
    free(inputC);
    fftw_destroy_plan(p);
    free(inputC_double);
    free(output_FFTW);


    return resFFT;
}

//vkfftPlanMany
VkFFTResult
vkfftPlanMany(VkGPU *vkGPU, VkFFTConfiguration configuration, VkFFTApplication appZ2Z, int rank, int *doubleComplex,
              int *doubleComplexPadded, int istride,
              int doubleComplexPaddedTotal, int *complexGridSizePadded, int ostride,
              int complexGridSizePaddedTotal, hipfftType type, int batch, hipStream_t *stream) {
    hipDoubleComplex *buffer = 0;
    hipDoubleComplex *buffer1 = 0;

    const int ZZ = 3, XX = 1, YY = 2;
    configuration.FFTdim = 3;
    configuration.size[0] = doubleComplex[ZZ];
    configuration.size[1] = doubleComplex[XX];
    configuration.size[2] = doubleComplex[YY];
    configuration.doublePrecision = true;
    //configuration.disableMergeSequencesR2C = 1;
    configuration.device = (hipDevice_t *) malloc(sizeof(hipDevice_t));
    hipError_t result = hipGetDevice(configuration.device);
    if (result != hipSuccess) {
        printf("VKFFT_ERROR_FAILED_TO_GET_DEVICE error: %d\n", result);
        return VKFFT_ERROR_FAILED_TO_GET_DEVICE;
    }
    configuration.num_streams = 1;
    configuration.stream = stream;

    uint64_t bufferSize =
            complexGridSizePadded[XX] * complexGridSizePadded[YY] * complexGridSizePadded[ZZ] *
            sizeof(hipfftDoubleComplex);
    std::cout << "bufferSize = " << bufferSize << std::endl;
    configuration.bufferSize = &bufferSize;
    configuration.bufferStride[0] = complexGridSizePadded[ZZ];
    configuration.bufferStride[1] = complexGridSizePadded[ZZ] * complexGridSizePadded[YY];
    configuration.bufferStride[2] = complexGridSizePadded[ZZ] * complexGridSizePadded[YY] * complexGridSizePadded[XX];
    result = hipMalloc((void **) &buffer, bufferSize);
    if (result != hipSuccess) return VKFFT_ERROR_FAILED_TO_ALLOCATE;
    configuration.buffer = (void **) &buffer;


    configuration.isInputFormatted = 1;
    configuration.inverseReturnToInputBuffer = 1;
    uint64_t inputBufferSize =
            doubleComplexPadded[XX] * doubleComplexPadded[YY] * doubleComplexPadded[ZZ] * sizeof(hipfftDoubleComplex);
    std::cout << "inputBufferSize = " << inputBufferSize << std::endl;
    configuration.inputBufferSize = &inputBufferSize;
    configuration.inputBufferStride[0] = doubleComplexPadded[ZZ];
    configuration.inputBufferStride[1] = doubleComplexPadded[ZZ] * doubleComplexPadded[YY];
    configuration.inputBufferStride[2] = doubleComplexPadded[ZZ] * doubleComplexPadded[YY] * doubleComplexPadded[XX];
    result = hipMemcpy(buffer, buffer1, inputBufferSize, hipMemcpyHostToDevice);
    if (result != hipSuccess) return VKFFT_ERROR_FAILED_TO_COPY;
    configuration.inputBuffer = (void **) &buffer1;
    VkFFTResult resFFT = initializeVkFFT(&appZ2Z, configuration);
    if (resFFT != VKFFT_SUCCESS) printf("VkFFT error: %d\n", resFFT);


    std::cout << "vkFFT: complex dim = " << doubleComplexPadded[XX] << "x" << doubleComplexPadded[YY] << "x"
              << doubleComplexPadded[ZZ] << std::endl;
    return resFFT;
}

VkFFTResult vkfftExecZ2Z(VkGPU *vkGPU, VkFFTApplication appZ2Z, VkFFTConfiguration configuration,
                         hipfftDoubleComplex *idata,
                         hipfftDoubleComplex *odata,
                         int direction, hipStream_t *stream) {
    uint64_t num_iter = (((uint64_t) 4096 * 1024.0 * 1024.0) / *configuration.bufferSize > 1000) ? 1000 :
                        (uint64_t) ((uint64_t) 4096 * 1024.0 * 1024.0) / *configuration.bufferSize;
    VkFFTLaunchParams launchParams = {};
    auto resFFT = performVulkanFFTiFFT(vkGPU, &appZ2Z, &launchParams, num_iter);
    return resFFT;
}

VkFFTResult vkfftBakeFFTPlan(interfaceFFTPlan* plan,hipStream_t *stream) {
    VkFFTResult res;
#if(__DEBUG__>0)
    printf("Begin initialization...\n");
#endif
    // If the plan was baked previously, the previous plan needs to be deleted
    if ((plan->app != NULL) && (plan->isBaked)) {
        deleteVkFFT(plan->app);
        plan->app = (VkFFTApplication*)calloc(1,sizeof(VkFFTApplication));
    }
    VkFFTConfiguration tmpConfig = *plan->config;
    res = initializeVkFFT(plan->app, tmpConfig);
#if(__DEBUG__>0)
    printf("    Done with initialization...\n");
#endif

    hipError_t cuda_res = hipSuccess;
    hipError_t cuda_res2 = hipSuccess;
    std::cout << "First Test" << std::endl;

    cuda_res = hipInit(0);
    if (cuda_res != hipSuccess) return VKFFT_ERROR_FAILED_TO_INITIALIZE;
    cuda_res2 = hipSetDevice((int) plan->device_id);
    if (cuda_res2 != hipSuccess) return VKFFT_ERROR_FAILED_TO_SET_DEVICE_ID;
    cuda_res = hipDeviceGet(&plan->device, (int) plan->device_id);
    if (cuda_res != hipSuccess) return VKFFT_ERROR_FAILED_TO_GET_DEVICE;
    cuda_res = hipCtxCreate(&plan->context, 0, (int) plan->device);
    if (cuda_res != hipSuccess) return VKFFT_ERROR_FAILED_TO_CREATE_CONTEXT;

    if (res == VKFFT_SUCCESS) {
        plan->isBaked = true;
    } else {
        plan->isBaked = false;
    }
    plan->notInit = true;
    return res;
}

int main() {
    interfaceFFTPlan* plan = {};
    hipStream_t *t;



    std::cout << "Second Test" << std::endl;
    printf("12 - VkFFT/FFTW C2C precision test in double precision\n");
    const int num_benchmark_samples = 1;
    const int num_runs = 1;

    uint64_t benchmark_dimensions[num_benchmark_samples][4] = {
            {(uint64_t) pow(2, 8), (uint64_t) pow(2, 8), (uint64_t) pow(2, 8), 3}
    };

    for (int n = 0; n < num_benchmark_samples; n++) {
        for (int r = 0; r < num_runs; r++) {
            get_VkFFT_double()
        }
    }
    std::cout << "Third Test" << std::endl;
    double *signal = (double *) malloc(sizeof(double) * 256 * 10);
    int dim_arry[3] = {1, 1, 1};
    for (long long int i = 0; i < 256 * 10; i++)
        signal[i] = i % 100;
    hipfftHandle plan;
    hipfftDoubleComplex *data;
    data = static_cast<hipfftDoubleComplex *>(malloc(sizeof(hipfftDoubleComplex) * 256 * 10));


    /* Create a 1D FFT plan. */
    hipfftPlanMany(&plan, 256, dim_arry, dim_arry, 1, 0,
                  dim_arry, 1, 0, HIPFFT_Z2Z, 10);
    /* Use the CUFFT plan to transform the signal in place. */
    hipfftExecZ2Z(plan, data, data, HIPFFT_FORWARD);
    /* Destroy the CUFFT plan. */
    std::cout << data[0].x << data[0].y << std::endl;

    hipfftDestroy(plan);
    hipFree(data);

    hipfftDoubleComplex *data1;
    data1 = static_cast<hipfftDoubleComplex *>(malloc(sizeof(hipfftDoubleComplex) * 256 * 10));

    for (long int i = 0; i < 256 * 10; i++) {
        data1[i].x = signal[i];
        data1[i].y = 0.;
    }
    VkFFTConfiguration configuration = {};
    VkFFTApplication appZ2Z = {};
    vkfftPlanMany(&vkGPU, configuration, appZ2Z, 256, dim_arry, dim_arry, 1, 0,
                  dim_arry, 1, 0, HIPFFT_Z2Z, 10, t);
    vkfftExecZ2Z(&vkGPU, appZ2Z, configuration, data1, data1, HIPFFT_FORWARD, t);

    std::cout << data1[1].x << data1[1].y << std::endl;

    return 0;
}
